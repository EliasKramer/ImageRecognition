#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t dot_product(int* c, const int* a, const int* b, unsigned int size);

__global__ void dot_product_cu(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}